#include "hip/hip_runtime.h"
#include "particleSystemGpu.h"

__constant__ float d_inv_masses[3];
__constant__ float d_charges[3];

__global__ void update_electrons(float timeDelta, int numParticles, int numE, int numP, float* positions, float* velocities, unsigned char* particleType) {

	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	if (gid < numE) {
		int part_type = particleType[gid];
		float force_x = 0.0;
		float force_y = 0.0;
		float force_z = 0.0;

		for (int j = 0; j < numE + numP; j++) {
			if (gid == j) {
				continue;
			}
			float dist_x = positions[gid * 4] - positions[j * 4];
			float dist_y = positions[gid * 4 + 1] - positions[j * 4 + 1];
			float dist_z = positions[gid * 4 + 2] - positions[j * 4 + 2];
			float dist_square = (dist_x * dist_x) + (dist_y * dist_y) + (dist_z * dist_z);
			float dist = sqrt(dist_square);

			float force = 0.0;
			//Coulomb force
			force += (float)coulomb_scalar / dist * d_charges[part_type] * d_charges[particleType[j]];

			//Break force into components
			force_x += force * dist_x / dist;
			force_y += force * dist_y / dist;
			force_z += force * dist_z / dist;
		}

		//Update velocities
		velocities[gid * 3] += force_x * d_inv_masses[part_type] * timeDelta;
		velocities[gid * 3 + 1] += force_y * d_inv_masses[part_type] * timeDelta;
		velocities[gid * 3 + 2] += force_z * d_inv_masses[part_type] * timeDelta;

		velocities[gid * 3] *= dampingFactor;
		velocities[gid * 3 + 1] *= dampingFactor;
		velocities[gid * 3 + 2] *= dampingFactor;
	}

}

__global__ void update_protons(float timeDelta, int numParticles, int numE, int numP, int numN, float* positions, float* velocities, unsigned char* particleType) {

	int gid = numE + blockIdx.x * blockDim.x + threadIdx.x;
	if (gid < numE + numP) {
		int part_type = particleType[gid];
		float force_x = 0.0;
		float force_y = 0.0;
		float force_z = 0.0;

		for (int j = 0; j < numE; j++) {
			float dist_x = positions[gid * 4] - positions[j * 4];
			float dist_y = positions[gid * 4 + 1] - positions[j * 4 + 1];
			float dist_z = positions[gid * 4 + 2] - positions[j * 4 + 2];
			float dist_square = (dist_x * dist_x) + (dist_y * dist_y) + (dist_z * dist_z);
			float dist = sqrt(dist_square);

			float force = (float)coulomb_scalar / dist * d_charges[part_type] * d_charges[particleType[j]];

			//Break force into components
			force_x += force * dist_x / dist;
			force_y += force * dist_y / dist;
			force_z += force * dist_z / dist;
		}


		for (int j = numE; j < numE + numP; j++) {
			if (gid == j) {
				continue;
			}
			float dist_x = positions[gid * 4] - positions[j * 4];
			float dist_y = positions[gid * 4 + 1] - positions[j * 4 + 1];
			float dist_z = positions[gid * 4 + 2] - positions[j * 4 + 2];
			float dist_square = (dist_x * dist_x) + (dist_y * dist_y) + (dist_z * dist_z);
			float dist = sqrt(dist_square);

			float force = (float)coulomb_scalar / dist * d_charges[part_type] * d_charges[particleType[j]];

			if (dist < yukawa_cutoff) {
				//force += yukawa_scalar * exp(-dist / yukawa_radius) / dist;
			}
			else {
				//force -= yukawa_scalar * exp(-dist / yukawa_radius) / dist;
			}

			//Break force into components
			force_x += force * dist_x / dist;
			force_y += force * dist_y / dist;
			force_z += force * dist_z / dist;

		}

		for (int j = numE + numP; j < numParticles; j++) {
			float dist_x = positions[gid * 4] - positions[j * 4];
			float dist_y = positions[gid * 4 + 1] - positions[j * 4 + 1];
			float dist_z = positions[gid * 4 + 2] - positions[j * 4 + 2];
			float dist_square = (dist_x * dist_x) + (dist_y * dist_y) + (dist_z * dist_z);
			float dist = sqrt(dist_square);

			float force = 0.0;

			if (dist < yukawa_cutoff) {
				force += yukawa_scalar * exp(-dist / yukawa_radius) / dist;
			}
			else {
				force -= yukawa_scalar * exp(-dist / yukawa_radius) / dist;
			}

			//Break force into components
			if (dist > 0) {
				force_x += force * dist_x / dist;
				force_y += force * dist_y / dist;
				force_z += force * dist_z / dist;
			}
		}

		//Update velocities
		velocities[gid * 3] += force_x * d_inv_masses[part_type] * timeDelta;
		velocities[gid * 3 + 1] += force_y * d_inv_masses[part_type] * timeDelta;
		velocities[gid * 3 + 2] += force_z * d_inv_masses[part_type] * timeDelta;

		velocities[gid * 3] *= dampingFactor;
		velocities[gid * 3 + 1] *= dampingFactor;
		velocities[gid * 3 + 2] *= dampingFactor;
	}
}


__global__ void update_neutrons(float timeDelta, int numParticles, int numE, int numP, int numN, float* positions, float* velocities, unsigned char* particleType) {
	int gid = numE + numP + blockIdx.x * blockDim.x + threadIdx.x;
	if (gid < numParticles) {
		int part_type = particleType[gid];
		float force_x = 0.0;
		float force_y = 0.0;
		float force_z = 0.0;

		for (int j = numE; j < numParticles; j++) {
			if (gid == j) {
				continue;
			}
			float dist_x = positions[gid * 4] - positions[j * 4];
			float dist_y = positions[gid * 4 + 1] - positions[j * 4 + 1];
			float dist_z = positions[gid * 4 + 2] - positions[j * 4 + 2];
			float dist_square = (dist_x * dist_x) + (dist_y * dist_y) + (dist_z * dist_z);
			float dist = sqrt(dist_square);

			//Strong Forces
			//P-N close attraction N-N close attraction 
			float force = 0.0f;
			if (part_type != 0 && particleType[j] != 0) {
				if (dist < yukawa_cutoff) {
					force += yukawa_scalar * exp(-dist / yukawa_radius) / dist;
				}
				else {
					force -= yukawa_scalar * exp(-dist / yukawa_radius) / dist;
				}

			}

			//Break force into components
			if (dist > 0) {
				force_x += force * dist_x / dist;
				force_y += force * dist_y / dist;
				force_z += force * dist_z / dist;
			}
			
		}
		//Update velocities
		velocities[gid * 3] += force_x * d_inv_masses[part_type] * timeDelta;
		velocities[gid * 3 + 1] += force_y * d_inv_masses[part_type] * timeDelta;
		velocities[gid * 3 + 2] += force_z * d_inv_masses[part_type] * timeDelta;

		velocities[gid * 3] *= dampingFactor;
		velocities[gid * 3 + 1] *= dampingFactor;
		velocities[gid * 3 + 2] *= dampingFactor;
	}

}

__global__ void update_naive(float timeDelta, int numParticles, float* positions, float* velocities, unsigned char* particleType) {
	
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	if (gid < numParticles) {
		int part_type = particleType[gid];
		float force_x = 0.0;
		float force_y = 0.0;
		float force_z = 0.0;
		for (int j = 0; j < numParticles; j++) {
			float dist_x = positions[gid * 4] - positions[j * 4];
			float dist_y = positions[gid * 4 + 1] - positions[j * 4 + 1];
			float dist_z = positions[gid * 4 + 2] - positions[j * 4 + 2];
			float dist_square = (dist_x * dist_x) + (dist_y * dist_y) + (dist_z * dist_z);
			float dist = sqrt(dist_square);
			if (gid == j) {
				continue;
			}
			float force = 0.0;
			//Coulomb force
			force += (float)coulomb_scalar / dist * d_charges[part_type] * d_charges[particleType[j]];



			//Strong Forces
			//P-N close attraction N-N close attraction 
			if (part_type != 0 && particleType[j] != 0) {
				if (dist < yukawa_cutoff) {
					force += yukawa_scalar * exp(-dist / yukawa_radius) / dist;
				}
				else {
					force -= yukawa_scalar * exp(-dist / yukawa_radius) / dist;
				}

			}
			//Break force into components
			force_x += force * dist_x / dist;
			force_y += force * dist_y / dist;
			force_z += force * dist_z / dist;
		}

		//Update velocities
		velocities[gid * 3] += force_x * d_inv_masses[part_type] * timeDelta;
		velocities[gid * 3 + 1] += force_y * d_inv_masses[part_type] * timeDelta;
		velocities[gid * 3 + 2] += force_z * d_inv_masses[part_type] * timeDelta;

		velocities[gid * 3] *= dampingFactor;
		velocities[gid * 3 + 1] *= dampingFactor;
		velocities[gid * 3 + 2] *= dampingFactor;
	}

}




__global__ void update_positions(float timeDelta, float * positions, float *velocities) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	//Update positions from velocities
	positions[gid * 4] += velocities[gid * 3] * timeDelta;
	if (abs(positions[gid * 4]) > boundingBox) {
		velocities[gid * 3] = -1 * velocities[gid * 3];
	}
		
	positions[gid * 4 + 1] += velocities[gid * 3 + 1] * timeDelta;
	if (abs(positions[gid * 4 + 1]) > boundingBox) {
		velocities[gid * 3 + 1] = -1 * velocities[gid * 3 + 1];
	}

	positions[gid * 4 + 2] += velocities[gid * 3 + 2] * timeDelta;
	if (abs(positions[gid * 4 + 2]) > boundingBox) {
		velocities[gid * 3 + 2] = -1 * velocities[gid * 3 + 2];
	}
}


__global__ void update_doubleBuffer(float timeDelta, int numParticles, float* src, float* dst, float* velocities, unsigned char* particleType) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	if (gid < numParticles) {
		int part_type = particleType[gid];
		float force_x = 0.0;
		float force_y = 0.0;
		float force_z = 0.0;
		for (int j = 0; j < numParticles; j++) {
			float dist_x = src[gid * 4] - src[j * 4];
			float dist_y = src[gid * 4 + 1] - src[j * 4 + 1];
			float dist_z = src[gid * 4 + 2] - src[j * 4 + 2];
			float dist_square = (dist_x * dist_x) + (dist_y * dist_y) + (dist_z * dist_z);
			float dist = sqrt(dist_square);
			if (gid == j) {
				continue;
			}
			float force = 0.0;
			//Coulomb force
			force += (float)coulomb_scalar / dist * d_charges[part_type] * d_charges[particleType[j]];



			//Strong Forces
			//P-N close attraction N-N close attraction 
			if (part_type != 0 && particleType[j] != 0) {
				if (dist < yukawa_cutoff) {
					force += yukawa_scalar * exp(-dist / yukawa_radius) / dist;
				}
				else {
					force -= yukawa_scalar * exp(-dist / yukawa_radius) / dist;
				}

			}
			//Break force into components
			force_x += force * dist_x / dist;
			force_y += force * dist_y / dist;
			force_z += force * dist_z / dist;
		}

		//Update velocities
		velocities[gid * 3] += force_x * d_inv_masses[part_type] * timeDelta;
		velocities[gid * 3 + 1] += force_y * d_inv_masses[part_type] * timeDelta;
		velocities[gid * 3 + 2] += force_z * d_inv_masses[part_type] * timeDelta;

		velocities[gid * 3] *= dampingFactor;
		velocities[gid * 3 + 1] *= dampingFactor;
		velocities[gid * 3 + 2] *= dampingFactor;

		dst[gid * 4] = src[gid * 4] + velocities[gid * 3] * timeDelta;
		if (abs(dst[gid * 4]) > boundingBox) {
			velocities[gid * 3] = -1 * velocities[gid * 3];
		}

		dst[gid * 4 + 1] = src[gid * 4 + 1] + velocities[gid * 3 + 1] * timeDelta;
		if (abs(dst[gid * 4 + 1]) > boundingBox) {
			velocities[gid * 3 + 1] = -1 * velocities[gid * 3 + 1];
		}

		dst[gid * 4 + 2] = src[gid * 4 + 2] +  velocities[gid * 3 + 2] * timeDelta;
		if (abs(dst[gid * 4 + 2]) > boundingBox) {
			velocities[gid * 3 + 2] = -1 * velocities[gid * 3 + 2];
		}
	}

}

__global__ void update_unroll(float timeDelta, int numParticles, float* positions, float* velocities, unsigned char* particleType) {
	int threadX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int threadY = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (threadX < numParticles && threadY < numParticles) {
		int part_type = particleType[threadX];
		float dist_x = positions[threadX * 4] - positions[threadY * 4];
		float dist_y = positions[threadX * 4 + 1] - positions[threadY * 4 + 1];
		float dist_z = positions[threadX * 4 + 2] - positions[threadY * 4 + 2];
		float dist_square = (dist_x * dist_x) + (dist_y * dist_y) + (dist_z * dist_z);
		float dist = sqrt(dist_square);
		if (threadX != threadY) {
			float force = 0.0;

			// Coulomb force
			force += (float)coulomb_scalar / dist * d_charges[part_type] * d_charges[particleType[threadY]];

			// Strong Forces
			// P-N close attraction N-N close attraction 
			if (part_type != 0 && particleType[threadY] != 0) {
				if (dist < yukawa_cutoff) {
					force += yukawa_scalar * exp(-dist / yukawa_radius) / dist;
				}
				else {
					force -= yukawa_scalar * exp(-dist / yukawa_radius) / dist;
				}
			}

			// Break force into components
			float force_x = force * dist_x / dist;
			float force_y = force * dist_y / dist;
			float force_z = force * dist_z / dist;

			// Update velocities
			atomicAdd(&velocities[threadX * 3], force_x * d_inv_masses[part_type] * timeDelta);
			atomicAdd(&velocities[threadX * 3 + 1], force_y * d_inv_masses[part_type] * timeDelta);
			atomicAdd(&velocities[threadX * 3 + 2], force_z * d_inv_masses[part_type] * timeDelta);
		}

		// Maybe add syncthreads here
		__syncthreads();

		// Apply damping
		if (threadX == threadY) {
			velocities[threadX * 3] *= dampingFactor;
			velocities[threadX * 3 + 1] *= dampingFactor;
			velocities[threadX * 3 + 2] *= dampingFactor;
		}
	}

}


ParticleSystemGPU::ParticleSystemGPU(int numParticles, int initMethod, int seed) {
		p_numParticles = numParticles;
		
#if (doubleBuffer)
		buf = false; //Note: this is used for doubleBuffering
#endif

#if (!UNROLL_ENABLE)
		dimBlock = TILE_SIZE;
		dimGrid = (int)ceil((float)numParticles / (float)TILE_SIZE);
#else
		dimBlock = dim3(TILE_SIZE, TILE_SIZE);
		dimGrid = dim3((int)ceil((float)numParticles / (float)TILE_SIZE), (int)ceil((float)numParticles / (float)TILE_SIZE));
#endif

		hipEventCreate(&event);

		// Initialize Positions array
		int positionElementsCount = 4 * numParticles;
		positions = new float[positionElementsCount];

		// Initialize Colors array
		int colorElementsCount = 3 * numParticles;
		colors = new unsigned int[colorElementsCount];

		int velocityElementsCount = 3 * numParticles;
		velocities = new float[velocityElementsCount];

		// Initialize Particle Type array
		particleType = new unsigned char[numParticles];

		// Circular initialization
		if (initMethod == 0) {
			for (unsigned int i = 0; i < numParticles; i++) {
				float theta = (float)((numParticles - 1 - i) / (float)numParticles * 2.0 * 3.1415); // Ensure floating-point division
				int pos_offset = 4;
			  int col_offset = 3;

				positions[i * pos_offset] = (float)cos(theta) * boundingBox;
				positions[i * pos_offset + 1] = (float)sin(theta) * boundingBox;
				positions[i * pos_offset + 2] = 1.0f * boundingBox;
				positions[i * pos_offset + 3] = 1.0f * boundingBox; // This will always stay as 1, it will be used for mapping 3D to 2D space

				colors[i * col_offset] = i % 255;
				colors[i * col_offset + 1] = 255 - (i % 255);
				colors[i * col_offset + 2] = 55;
			}
		}
		//Hydrogen atoms
		else if (initMethod == 1) {
			if (seed != -1) {
				srand(seed);
			}
			int it = numParticles / 3;
			int pos_offset = 4;
			int vel_offset = 3;
			numProtons = it;
			numNeutrons = it;
			numElectrons = numParticles - 2 * it;
			for (unsigned int i = numElectrons; i < numElectrons + numProtons; i++) {

				//Pair up protons and neutrons
				float pos_X = ((float)(rand() % 2000) - 1000.0) / 1000.0 * boundingBox;
				float pos_Y = ((float)(rand() % 2000) - 1000.0) / 1000.0 * boundingBox;
				float pos_Z = ((float)(rand() % 2000) - 1000.0) / 1000.0 * boundingBox;

				positions[i * pos_offset] = pos_X;
				positions[(i + it) * pos_offset] = (float)(pos_X + yukawa_radius);

				positions[i * pos_offset + 1] = pos_Y;
				positions[(i + it) * pos_offset + 1] = (float)(pos_Y + yukawa_radius);

				positions[i * pos_offset + 2] = pos_Z;
				positions[(i + it) * pos_offset + 2] = (float)(pos_Z + yukawa_radius);

				particleType[i] = 1;
				particleType[i + it] = 2;
			}
			//Scatter in some electrons
			for (unsigned int i = 0; i < numElectrons; i++) {
				positions[i * pos_offset] = ((float)(rand() % 2000) - 1000.0) / 1000.0 * boundingBox;
				positions[i * pos_offset + 1] = ((float)(rand() % 2000) - 1000.0) / 1000.0 * boundingBox;
				positions[i * pos_offset + 2] = ((float)(rand() % 2000) - 1000.0) / 1000.0 * boundingBox;

				particleType[i] = 0;
			}

			//Initialize velocities to 0 and give particles the proper color.
			for (unsigned int i = 0; i < numParticles; i++) {

				positions[i * pos_offset + 3] = 1.0f * boundingBox; // This will always stay as 1, it will be used for mapping 3D to 2D space

				velocities[i * vel_offset] = 0;
				velocities[i * vel_offset + 1] = 0;
				velocities[i * vel_offset + 2] = 0;

				// Sets color based on particle type
				if (particleType[i] == 0) { // If Electron
					colors[i * vel_offset] = ELECTRON_COLOR[0];
					colors[i * vel_offset + 1] = ELECTRON_COLOR[1];
					colors[i * vel_offset + 2] = ELECTRON_COLOR[2];
				}
				else if (particleType[i] == 1) { // If Proton
					colors[i * vel_offset] = PROTON_COLOR[0];
					colors[i * vel_offset + 1] = PROTON_COLOR[1];
					colors[i * vel_offset + 2] = PROTON_COLOR[2];
				}
				else {
					colors[i * vel_offset] = NEUTRON_COLOR[0]; //Else neutron
					colors[i * vel_offset + 1] = NEUTRON_COLOR[1];
					colors[i * vel_offset + 2] = NEUTRON_COLOR[2];
				}

			}
		}
		// Random initialization in 3 dimensions
		else if (initMethod == 2) {
			if (seed != -1) {
				srand(seed);
			}
			
#if (orderedParticles)
			int it = numParticles / 3;
			numProtons = it;
			numNeutrons = it;
			numElectrons = numParticles - 2 * it;
			std::cout << "numProtons" << numProtons << std::endl;
			std::cout << "numNeutrons" << numNeutrons << std::endl;
			std::cout << "numElectrons" << numElectrons << std::endl;

			electronGridSize = (int)ceil((float)numElectrons / (float)TILE_SIZE);
			protonGridSize = (int)ceil((float)numProtons / (float)TILE_SIZE);
			neutronGridSize = (int)ceil((float)numNeutrons / (float)TILE_SIZE);
#else
			numProtons = 0;
			numNeutrons = 0;
			numElectrons = 0;
#endif
			for (unsigned int i = 0; i < numParticles; i++) {
				int pos_offset = 4;
				int vel_offset = 3;
				// Randomly initialize position in range [-1,1)
				positions[i * pos_offset] = ((float)(rand() % 2000) - 1000.0) / 1000.0 * boundingBox;
				positions[i * pos_offset + 1] = ((float)(rand() % 2000) - 1000.0) / 1000.0 * boundingBox;
				positions[i * pos_offset + 2 ] = ((float)(rand() % 2000) - 1000.0) / 1000.0 * boundingBox;
				positions[i * pos_offset + 3 ] = 1.0f * boundingBox; // This will always stay as 1, it will be used for mapping 3D to 2D space

				// Randomly initializes velocity in range [-250000,250000)
				velocities[i * vel_offset] = ((float)(rand() % 500) - 250.0) * 1000.0;
				velocities[i * vel_offset + 1] = ((float)(rand() % 500) - 250.0) * 1000.0;
				velocities[i * vel_offset + 2] = ((float)(rand() % 500) - 250.0) * 1000.0;

				// Generates random number (either 0, 1, 2) from uniform dist
#if (orderedParticles)
				if (i < numElectrons) {
					particleType[i] = 0;
				} 
				else if (i < numElectrons + numProtons) {
					particleType[i] = 1;
				}
				else {
					particleType[i] = 2;
				}
#else
				int type = rand() % 3;
				particleType[i] = type;
				if (type == 0) {
					numElectrons++;
				}
				else if (type == 1) {
					numProtons++;
				}
				else {
					numNeutrons++;
				}

#endif

				// Sets color based on particle type
				if (particleType[i] == 0) { // If Electron
					colors[i * vel_offset] = ELECTRON_COLOR[0];
					colors[i * vel_offset + 1] = ELECTRON_COLOR[1];
					colors[i * vel_offset + 2] = ELECTRON_COLOR[2];
				}
				else if (particleType[i] == 1) { // If Proton
					colors[i * vel_offset] = PROTON_COLOR[0];
					colors[i * vel_offset + 1] = PROTON_COLOR[1];
					colors[i * vel_offset + 2] = PROTON_COLOR[2];
				}
				else {
					colors[i * vel_offset] = NEUTRON_COLOR[0]; //Else neutron
					colors[i * vel_offset + 1] = NEUTRON_COLOR[1];
					colors[i * vel_offset + 2] = NEUTRON_COLOR[2];
				}
			}
		}
		//Error bad method
		else {
			std::cerr << "Bad Initialization";
		}

#if (RENDER_ENABLE)
			glGenVertexArrays(1, &VAO);

			glBindVertexArray(VAO);

			glGenBuffers(1, &positionBuffer);
			glGenBuffers(1, &colorBuffer);

			glBindBuffer(GL_ARRAY_BUFFER, positionBuffer);
			glBufferData(GL_ARRAY_BUFFER, sizeof(float) * 4 * numParticles, positions, GL_STREAM_DRAW);
			glVertexAttribPointer(0, 4, GL_FLOAT, GL_FALSE, 0, (void*)0);

			glBindBuffer(GL_ARRAY_BUFFER, colorBuffer);
			glBufferData(GL_ARRAY_BUFFER, sizeof(unsigned int) * 3 * numParticles, colors, GL_STATIC_DRAW);
			glVertexAttribIPointer(1, 3, GL_UNSIGNED_INT, 0, (void*)0);

			glBindBuffer(GL_ARRAY_BUFFER, 0);

			glEnableVertexAttribArray(0);
			glEnableVertexAttribArray(1);

			shaderProgram = new Shader();
#endif

		//Initialize device

		hipMemcpyToSymbol(HIP_SYMBOL(d_inv_masses), inv_masses, 3 * sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(d_charges), charges, 3 * sizeof(float));

#if (RENDER_ENABLE)
		hipGraphicsGLRegisterBuffer(&positionResource, positionBuffer, cudaGraphicsMapFlagsNone);
#else
		hipMalloc(&d_positions, positionElementsCount * sizeof(float));
		hipMemcpy(d_positions, positions, positionElementsCount * sizeof(float), hipMemcpyHostToDevice);
#if (doubleBuffer)
		
		hipMalloc(&d_positions2, positionElementsCount * sizeof(float));
		hipMemcpy(d_positions2, positions, positionElementsCount * sizeof(float), hipMemcpyHostToDevice);
		src = d_positions;
		dst = d_positions2;
		
#endif

#endif

		hipMalloc(&d_velocities, velocityElementsCount * sizeof(float));
		hipMemcpy(d_velocities, velocities, velocityElementsCount * sizeof(float), hipMemcpyHostToDevice);

#if (RENDER_ENABLE)
		hipGraphicsGLRegisterBuffer(&colorResource, colorBuffer, cudaGraphicsMapFlagsNone);
#else
		hipMalloc(&d_colors, colorElementsCount * sizeof(unsigned int));
		hipMemcpy(d_colors, colors, colorElementsCount * sizeof(unsigned int), hipMemcpyHostToDevice);
#endif
	
		hipMalloc(&d_particleType, numParticles * sizeof(unsigned char));
		hipMemcpy(d_particleType, particleType, numParticles * sizeof(unsigned char), hipMemcpyHostToDevice);

#if binningGPU
		int binSize = binX * binY * binZ * binDepth
		hipMalloc(&d_bin, binSize * sizeof(int);
		int dimBlock = 256;
		int dimGrid = (int)ceil((float)binSize / (float)dimBlock);
		initializeBuffer<<<dimGrid, dimBlock>>>(binSize, d_bin);
		
		
		hipMalloc(&d_overflow, overflowSize * sizeof(int));
		dimGrid = (int)ceil((float)overflowSize / (float)dimBlock);
		initializeBuffer<<<dimGrid, dimBlock>>>(overflowSize, d_overflow);
#endif
}

float* ParticleSystemGPU::getPositions() {
#if (RENDER_ENABLE)
		size_t Size;
		hipGraphicsMapResources(1, &positionResource, 0);
		hipGraphicsResourceGetMappedPointer((void**)&d_positions, &Size, positionResource);
#endif

		int numBytes = p_numParticles * 4 * sizeof(float);
#if doubleBuffer
		if (buf) {
			hipMemcpy(positions, d_positions2, numBytes, hipMemcpyDeviceToHost);
		}
		else {
			hipMemcpy(positions, d_positions, numBytes, hipMemcpyDeviceToHost);
		}
#else
		hipMemcpy(positions, d_positions, numBytes, hipMemcpyDeviceToHost);
#endif
		

#if (RENDER_ENABLE)
		hipGraphicsUnmapResources(1, &positionResource, 0);
#endif
		return positions;
}

float* ParticleSystemGPU::getVelocities() {
		int numBytes = p_numParticles * 3 * sizeof(float);
		hipMemcpy(velocities, d_velocities, numBytes, hipMemcpyDeviceToHost);
		return velocities;
}

unsigned int* ParticleSystemGPU::getColors() {
#if (RENDER_ENABLE)
		size_t Size;
		hipGraphicsMapResources(1, &colorResource, 0);
		hipGraphicsResourceGetMappedPointer((void**)&d_colors, &Size, colorResource);
#endif
	
		int numBytes = p_numParticles * 3 * sizeof(unsigned int);
		hipMemcpy(colors, d_colors, numBytes, hipMemcpyDeviceToHost);

#if (RENDER_ENABLE)
		hipGraphicsUnmapResources(1, &colorResource, 0);
#endif
	return colors;
}



void ParticleSystemGPU::update(float timeDelta) {
#if (RENDER_ENABLE)
		size_t Size;
		hipGraphicsMapResources(1, &positionResource, 0);
		hipGraphicsResourceGetMappedPointer((void**)&d_positions, &Size, positionResource);
#endif
#if (doubleBuffer)
		update_doubleBuffer<<<dimGrid, dimBlock>>>(timeDelta, p_numParticles, src, dst, d_velocities, d_particleType);
#elif (orderedParticles)
		update_electrons<<<electronGridSize, dimBlock>>>(timeDelta, p_numParticles, numElectrons, numProtons, d_positions, d_velocities, d_particleType);
		update_protons<<<protonGridSize, dimBlock>>>(timeDelta, p_numParticles, numElectrons, numProtons, numNeutrons, d_positions, d_velocities, d_particleType);
		update_neutrons<<<neutronGridSize, dimBlock>>>(timeDelta, p_numParticles, numElectrons, numProtons, numNeutrons, d_positions, d_velocities, d_particleType);

		update_positions<<<dimGrid, dimBlock>>>(timeDelta, d_positions, d_velocities);
#elif (UNROLL_ENABLE)
		update_unroll<<<dimGrid, dimBlock>>>(timeDelta, p_numParticles, d_positions, d_velocities, d_particleType);
		//hipDeviceSynchronize();
		update_positions<<<dimGrid, dimBlock>>>(timeDelta, d_positions, d_velocities);
#else
		update_naive<<<dimGrid, dimBlock>>>(timeDelta, p_numParticles, d_positions, d_velocities, d_particleType);

		update_positions<<<dimGrid, dimBlock>>>(timeDelta, d_positions, d_velocities);
		//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
#endif

		hipError_t cudaStatusFlag = hipGetLastError();
		if (cudaStatusFlag != hipSuccess) {
			std::cerr << "Kernel failed: " << hipGetErrorString(cudaStatusFlag) << std::endl;

			// Do hipFree here

			//return false;

			// Should probably make this function return a boolean which indicates success
			// Stop sim if error encountered
		}

		hipEventRecord(event);

		hipEventSynchronize(event);

#if (RENDER_ENABLE)
		hipGraphicsUnmapResources(1, &positionResource, 0);
#endif
}

void ParticleSystemGPU::flip() {
	float* temp;
	temp = dst;
	dst = src;
	src = temp;
	buf = !buf;
}



void ParticleSystemGPU::writecurpostofile(char* file, int steps, float milliseconds) {
		getPositions();
		std::ofstream outfile(file);

		if (outfile.is_open()) {
			outfile << "particles:" << p_numParticles << " iterations:" << steps << " timing:" << milliseconds << " doubleBuffer:" << doubleBuffer << "\n";
			for (int i = 0; i < p_numParticles; i++) {
#if (STORAGE_TYPE && !RENDER_ENABLE)
				outfile << positions[i] << " ";
				outfile << positions[i + p_numParticles] << " ";
				outfile << positions[i + 2 * p_numParticles] << " ";
				outfile << positions[i + 3 * p_numParticles] << "\n";
#else
				outfile << positions[i * 4] << " ";
				outfile << positions[i * 4 + 1] << " ";
				outfile << positions[i * 4 + 2] << " ";
				outfile << positions[i * 4 + 3] << "\n";
#endif
			}
		}
		else {
			std::cerr << "Unable to open file: " << file << std::endl;
		}
}

	
void ParticleSystemGPU::display() {
#if (RENDER_ENABLE)
		//Positions are already updated since we work directly on the data!

		glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		shaderProgram->Activate();

		glPointSize(2.0);
		
		glDrawArrays(GL_POINTS, 0, p_numParticles);
#endif
}

ParticleSystemGPU::~ParticleSystemGPU() {
	p_numParticles = 0;
	delete[] positions;
	delete[] colors;
	delete[] velocities;
	delete[] particleType;

	//VBO will handle positions and colors buffers if we rendered.
	hipFree(d_velocities);
	hipFree(d_particleType);

	hipEventDestroy(event);
#if (RENDER_ENABLE)
		delete shaderProgram;
		glDeleteVertexArrays(1, &VAO);
		glDeleteBuffers(1, &positionBuffer);
		glDeleteBuffers(1, &colorBuffer);
#else
		hipFree(d_positions);
		hipFree(d_colors);
#if (doubleBuffer)
		hipFree(d_positions2);
#endif
#endif
	
}