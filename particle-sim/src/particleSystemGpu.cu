#include "hip/hip_runtime.h"
#include "particleSystemGpu.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <fstream>


#define TILE_SIZE 128

__constant__ double inv_masses[3];
__constant__ float charges[3];


__global__ void update_naive(float timeDelta, int numParticles, float coulomb_scalar, float yukawa_scalar, float yukawa_radius, float yukawa_cutoff, float* positions, float* velocities, unsigned char* particleType) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	int part_type = particleType[gid];
	double force_x = 0.0f;
	double force_y = 0.0f;
	double force_z = 0.0f;
	for (int j = 0; j < numParticles; j++) {
		float dist_square = (positions[gid] - positions[j]) * (positions[gid] - positions[j]) + (positions[gid] - positions[j + 1]) * (positions[gid] - positions[j + 1]);
		float dist = sqrt(dist_square);
		if (gid == j || dist < yukawa_cutoff) {
			continue;
		}

		double force = (double)coulomb_scalar / dist_square * charges[part_type] * charges[particleType[j]];
		double dist_x = (double)positions[gid] - positions[j];
		double dist_y = (double)positions[gid + 1] - positions[j + 1];
		force_x += force * dist_x / dist;
		force_y += force * dist_y / dist;

		//Strong Forces
		//P-N close attraction N-N close attraction 
		if (part_type != 0 && particleType[j] != 0) {
			force = yukawa_scalar * exp(dist / yukawa_radius) / dist;
			force_x += force * dist_x / dist;
			force_y += force * dist_y / dist;
		}

		//Update velocities
		velocities[gid] += force_x * inv_masses[part_type] * 1e-9 * timeDelta;
		velocities[gid + 1] += force_y * inv_masses[part_type] * 1e-9 * timeDelta;
		velocities[gid + 2] += force_z * inv_masses[part_type] * 1e-9 * timeDelta;
		
		//Update positions from velocities
		positions[gid * 4] += velocities[gid * 3];
		if (abs(positions[gid * 4]) > 1) {
			velocities[gid * 3] = -1 * velocities[gid * 3];
		}
		positions[gid * 4 + 1] += velocities[gid * 3 + 1];
		if (abs(positions[gid * 4 + 1]) > 1) {
			velocities[gid * 3 + 1] = -1 * velocities[gid * 3 + 1];
		}
		positions[gid * 4 + 2] += velocities[gid * 3 + 2];
		if (abs(positions[gid * 4 + 2]) > 1) {
			velocities[gid * 3 + 2] = -1 * velocities[gid * 3 + 2];
		}
	}


}



ParticleSystemGPU::ParticleSystemGPU(int numParticles, int initMethod, int seed, bool render) {
	p_numParticles = numParticles;
	p_render = render;
	blockSize = TILE_SIZE;
	gridSize = (int)ceil((float)numParticles / (float)TILE_SIZE);

	


	// Initialize Positions array
	int positionElementsCount = 4 * numParticles;
	positions = new float[positionElementsCount];
	//memset(positions, 0, positionElementsCount);

	// Initialize Colors array
	int colorElementsCount = 3 * numParticles;
	colors = new unsigned int[colorElementsCount];
	//memset(colors, 0, colorElementsCount);

	int velocityElementsCount = 3 * numParticles;
	velocities = new float[velocityElementsCount];

	// Initialize Particle Type array
	particleType = new unsigned char[numParticles];

	coulomb_scalar = 2.310272969e-4; //N*nanometers^2
	yukawa_scalar = 1.9692204e-3;    //Experimentally obtained
	yukawa_radius = 1.4e-3;			 //Radius of strength.
	yukawa_cutoff = 1e-3;          //Sweet spot. (Strong force likes to be between 0.8 and 1.4 fm.

	// Circular initialization
	if (initMethod == 0) {
		for (unsigned int i = 0; i < numParticles; i++) {
			float theta = (float)((numParticles - 1 - i) / (float)numParticles * 2.0 * 3.1415); // Ensure floating-point division
			positions[i * 4] = (float)cos(theta);
			positions[i * 4 + 1] = (float)sin(theta);
			positions[i * 4 + 2] = 1.0f;
			positions[i * 4 + 3] = 1.0f; // This will always stay as 1, it will be used for mapping 3D to 2D space

			colors[i * 3] = i % 255;
			colors[i * 3 + 1] = 255 - (i % 255);
			colors[i * 3 + 2] = 55;
		}

	}
	//Read from a file
	else if (initMethod == 1) {

	}
	// Random initialization in 3 dimensions
	else if (initMethod == 2) {
		if (seed != -1) {
			srand(seed);
		}
		for (unsigned int i = 0; i < numParticles; i++) {
			// Randomly initialize position in range [-1,1)
			positions[i * 4] = ((float)(rand() % 2000) - 1000.0) / 1000.0;
			positions[i * 4 + 1] = ((float)(rand() % 2000) - 1000.0) / 1000.0;
			positions[i * 4 + 2] = ((float)(rand() % 2000) - 1000.0) / 1000.0;
			positions[i * 4 + 3] = 1.0f; // This will always stay as 1, it will be used for mapping 3D to 2D space

			// Randomly initializes velocity in range [-0.0025,0.0025)
			velocities[i * 3] = ((float)(rand() % 500) - 250.0) / 100000.0;
			velocities[i * 3 + 1] = ((float)(rand() % 500) - 250.0) / 100000.0;
			velocities[i * 3 + 2] = ((float)(rand() % 500) - 250.0) / 100000.0;

			// Generates random number (either 0, 1, 2) from uniform dist
			//particleType[i] = rand() % 3 % 2; 
			particleType[i] = rand() % 3;

			// Sets color based on particle type
			if (particleType[i] == 0) { // If Electron
				colors[i * 3] = 0;
				colors[i * 3 + 1] = 180;
				colors[i * 3 + 2] = 255;

			}
			else if (particleType[i] == 1) { // If Proton
				colors[i * 3] = 255;
				colors[i * 3 + 1] = 0;
				colors[i * 3 + 2] = 0;
			}
			else {
				colors[i * 3] = 255; //Else neutron
				colors[i * 3 + 1] = 0;
				colors[i * 3 + 2] = 180;

			}
		}
	}
	//Error bad method
	else {

	}

	//Initialize device

	double inv_mass[] = { 1.09776e30, 5.978638e26, 5.978638e26 };
	float charge[] = { -1, 1, 0 };

	hipMemcpyToSymbol(HIP_SYMBOL(inv_masses), inv_mass, 3 * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(charges), charge, 3 * sizeof(float));


	if (!render) {
		hipMalloc(&d_positions, positionElementsCount * sizeof(float));
		hipMemcpy(d_positions, velocities, velocityElementsCount * sizeof(float), hipMemcpyHostToDevice);
	}
	//p_buffer->mapPositions(d_positions);
	//hipMemcpy(d_positions, positions, positionElementsCount * sizeof(float), hipMemcpyHostToDevice);
	//p_buffer->unmapPositions();


	hipMalloc(&d_velocities, velocityElementsCount * sizeof(float));
	hipMemcpy(d_velocities, velocities, velocityElementsCount * sizeof(float), hipMemcpyHostToDevice);

	if (!render) {
		hipMalloc(&d_colors, colorElementsCount * sizeof(unsigned int));
		hipMemcpy(d_colors, colors, colorElementsCount * sizeof(unsigned int), hipMemcpyHostToDevice);
	}
	//p_buffer->mapColors(d_colors);
	//hipMemcpy(d_colors, colors, colorElementsCount * sizeof(float), hipMemcpyHostToDevice);
	//p_buffer->unmapPositions();
	
	
	hipMalloc(&d_particleType, numParticles * sizeof(unsigned char));
	hipMemcpy(d_particleType, particleType, numParticles * sizeof(unsigned char), hipMemcpyHostToDevice);
}

//This should be run before any other functions. (The construction is dependent on this running.
void ParticleSystemGPU::assignBuffer(Buffer* buffer) {
	p_buffer = buffer;
}

float* ParticleSystemGPU::getPositions() {
	if (p_render) {
		p_buffer->mapPositions(d_positions);
	}
	
	int numBytes = p_numParticles * 4 * sizeof(float);
	hipMemcpy(d_positions, positions, numBytes, hipMemcpyDeviceToHost);
	if (p_render) {
		p_buffer->unmapPositions();
	}
	
	return positions;
}

float* ParticleSystemGPU::getVelocities() {
	int numBytes = p_numParticles * 3 * sizeof(float);
	hipMemcpy(d_velocities, velocities, numBytes, hipMemcpyDeviceToHost);
	return velocities;
}

unsigned int* ParticleSystemGPU::getColors() {
	if (p_render) {
		p_buffer->mapColors(d_colors);
	}
	
	int numBytes = p_numParticles * 3 * sizeof(unsigned int);
	hipMemcpy(d_colors, colors, numBytes, hipMemcpyDeviceToHost);
	if (p_render) {
		p_buffer->unmapColors();
	}
	
	return colors;
}



void ParticleSystemGPU::update(float timeDelta) {
	update_naive<<<gridSize, blockSize>>>(timeDelta, p_numParticles, coulomb_scalar, yukawa_scalar, yukawa_radius, yukawa_cutoff, positions, velocities, particleType);
}



void ParticleSystemGPU::writecurpostofile(char* file) {
	getPositions();
	std::ofstream outfile(file);

	if (outfile.is_open()) {
		for (int i = 0; i < p_numParticles; i++) {
			outfile << positions[i * 4] << " ";
			outfile << positions[i * 4 + 1] << " ";
			outfile << positions[i * 4 + 2] << " ";
			outfile << positions[i * 4 + 3] << "\n";
		}
	}
	else {
		std::cerr << "Unable to open file: " << file << std::endl;
	}
}

	


ParticleSystemGPU::~ParticleSystemGPU() {
	p_numParticles = 0;
	p_render = 0;
	delete[] positions;
	delete[] colors;
	delete[] velocities;
	delete[] particleType;

	//VBO will handle positions and colors buffers if we rendered.
	hipFree(d_velocities);
	hipFree(d_particleType);
	if (!p_render) {
		hipFree(d_positions);
		hipFree(d_colors);
	}
	
}